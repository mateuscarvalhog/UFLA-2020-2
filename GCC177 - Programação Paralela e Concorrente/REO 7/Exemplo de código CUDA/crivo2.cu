#include "hip/hip_runtime.h"
//Rodrigo Amamdor Coelho: 28/02/2011

//Este código é o Crivo de Aristóteles executado paralelamente na GPU.
//
//


#include <hip/hip_runtime.h>
#include <cutil.h>
#include<stdio.h>
#include <hip/hip_runtime.h> 



///////////////////////////////////////////////////////////////////////////////////
//GPU KERNEL

__shared__ int k;
__global__ void crivo_array(int *a)

{

	int idx = blockIdx.x*blockDim.x+threadIdx.x; // parassar entre as threads e blocls de threads
	a[idx]= idx;
	k=2;


	while(k*2<=idx){

		if (((a[idx] % k) == (0))&&(a[idx] > 0)&&(a[idx] != k)){ //verifico se é multipo de k
			a[idx] = a[idx] * -1; //marco o elemento na lista
		}

		k++;

		while(a[k]<0){
			k++;
		}

	}

}

///////////////////////////////////////////////////////////////////////////////////
//CPU

 int main(void)
{

	int *a_h, *a_d, threads, blocks;

	const int N=100000; //Numero de elementos da lista


	//Determino o numero de threads e blocks de threads usados
	if (N < 256)
	{
		threads = N;
		blocks = 1;
	}
	else
	{
		threads = 256;
		if ((N % threads) == 0)
			blocks = N/threads;
		else
			blocks = (N/threads) + 1;
	}
	//Fim do determincao	


	size_t size = N*sizeof(int); //determinando o tamanho do vetor de inteiros

	a_h=(int*)malloc(size); //alocando espaço na memoria da CPU

	hipMalloc((void**)&a_d,size); //alocando espaço na memoria da GPU



	unsigned int timer = 0;
	CUT_SAFE_CALL(cutCreateTimer(&timer)); //cria função de calculo de tempo no CUDA
	CUT_SAFE_CALL(cutStartTimer(timer)); //inicia função de calculo de tempo no CUDA



	//faz crivo
	hipMemcpy(a_d,a_h,size,hipMemcpyHostToDevice);
	crivo_array<<<blocks,threads>>>(a_d);
	hipMemcpy(a_h,a_d,size,hipMemcpyDeviceToHost);
	//fim do crivo




	printf("\nTempo de computacao: %f (ms) \n\n", cutGetTimerValue(timer)); 
	CUT_SAFE_CALL(cutDeleteTimer(timer)); //finaliza com a função de contagem de tempo
/*
	printf("São primos os numeros entre 2 e %d\n", N);
	for(int i=2; i<N; i++)
		if (a_h[i] > 0)
			printf("%d\n",a_h[i]); //imprimo somente os numeros não marcados (primos)
*/

	free(a_h); // Liberando memoria da CPU
	hipFree(a_d); // Liberando memoria da GPU

}
