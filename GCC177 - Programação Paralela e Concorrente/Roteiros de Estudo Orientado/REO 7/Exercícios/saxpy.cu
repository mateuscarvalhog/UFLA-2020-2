
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

int main() {
    int N = 512;
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float)); 
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 0.5;
        y[i] = 2.0;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    int nBlocks = (N + 255) / 256;
    saxpy<<<nBlocks, 256>>>(N, 2.0, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Final array: ");
    for(int i = 0; i < N; i++) {
        printf("%f ", y[i]);
    }
    printf("\n");

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}